#include "dllexport.h"
#include <moderngpu/context.hxx>
#include <moderngpu/kernel_segsort.hxx>
#include <stdint.h>

namespace {

template <class key_t, class val_t>
void segsortpairs(key_t *d_keys, val_t *d_vals, uint32_t n_vals,
                  const int32_t *d_segments, uint32_t n_segs,
                  hipStream_t stream) {

  mgpu::standard_context_t context;
  mgpu::segmented_sort(d_keys, d_vals, n_vals, d_segments, n_segs,
                       mgpu::less_t<key_t>(), context);
}

} // namespace

extern "C" {

#define WRAP(F, Tkey, Tval)                                                    \
  DLLEXPORT void segsortpairs_##F(Tkey *d_keys, Tval *d_vals, unsigned N,      \
                                  const int *d_segments, unsigned NumSegs,     \
                                  hipStream_t stream) {                       \
    segsortpairs(d_keys, d_vals, N, d_segments, NumSegs, stream);              \
  }

WRAP(int32, int32_t, unsigned)
WRAP(int64, int64_t, unsigned)
WRAP(uint32, uint32_t, unsigned)
WRAP(uint64, uint64_t, unsigned)
WRAP(float32, float, unsigned)
WRAP(float64, double, unsigned)
}
