#include <stdint.h>
#include <moderngpu/context.hxx>
#include <moderngpu/kernel_segsort.hxx>
#include "dllexport.h"

namespace mgpu{
	std::string stringprintf(const char* format, ...) { return std::string(); }
}

namespace {

template<class Tkey, class Tval>
void segsortpairs(Tkey *d_keys,
  Tval *d_vals,
  int N,
  const int *d_segments,
  unsigned NumSegs,
  hipStream_t stream)
{

  mgpu::standard_context_t context;
  mgpu::segmented_sort(d_keys, d_vals, N, d_segments, NumSegs, mgpu::less_t<Tkey>(), context);

}

} // end static namespace


extern "C" {

#define WRAP(F, Tkey, Tval)												\
DLLEXPORT void segsortpairs_##F( Tkey *d_keys,                          \
					   Tval *d_vals,									\
					   unsigned N,										\
					   const int *d_segments,							\
					   unsigned NumSegs,								\
					   hipStream_t stream	)							\
{  segsortpairs(d_keys, d_vals, N, d_segments, NumSegs, stream);  }

WRAP(int32, int32_t, unsigned)
WRAP(int64, int64_t, unsigned)
WRAP(uint32, uint32_t, unsigned)
WRAP(uint64, uint64_t, unsigned)
WRAP(float32, float, unsigned)
WRAP(float64, double, unsigned)


}
